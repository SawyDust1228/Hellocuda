#include "hip/hip_runtime.h"

#include "kernel.cuh"
#include <hipfft/hipfft.h>

// #define DEBUG

#define CONST_MEMORY 25
#define MAX_NODES 1000
__constant__ float weight[CONST_MEMORY];
__constant__ node nodes[MAX_NODES];

__global__ void vector_add_device(float* v1, float* v2, float* result, int n) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id < n) {
        result[id] = v1[id] + v2[id];
        // printf("%f", result[id]);
    }
}

extern "C"
void vector_add(float* v1, float* v2, float* result, int n) {
    int size = sizeof(float) * n;
    float* v1_gpu; float* v2_gpu; float* result_gpu;
    hipMalloc((void**) &v1_gpu, size);
    hipMalloc((void**) &v2_gpu, size);
    hipMalloc((void**) &result_gpu, size);
    hipMemcpy(v1_gpu, v1, size, hipMemcpyHostToDevice);
    hipMemcpy(v2_gpu, v2, size, hipMemcpyHostToDevice);

    vector_add_device<<<ceil(n / 256.0), 256>>>(v1_gpu, v2_gpu, result_gpu, n);
    hipMemcpy(result, result_gpu, size, hipMemcpyDeviceToHost);
    hipFree(v1_gpu);hipFree(v2_gpu);hipFree(result_gpu);
}

__device__ bool is_valid(int id, int w, int h) {
    return (id > -1 && id < w * h) ? true : false;
}

__global__ void blur_kernel(float* matrix, float* result, int w, int h, int window) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int idy = blockDim.y * blockIdx.y + threadIdx.y;
    int step = (window - 1) / 2;
    int id = idx * w + idy;
    if(!is_valid(id, w, h)) {
        return;
    }
    float value = 0;
    for(int i = -step; i <= step; i++) {
        for(int j = -step; j <= step; j++) {
            int id_window = (idx + i) * w + (idy + j);
            if((idx + i) > -1 && (idx + i) < h && (idy + j) > -1 && (idy + j) < w) {
                value += matrix[id_window];
            }
        }
    }
    result[id] = value;
    // printf("Idx : %d, Idy : %d, value : %.2f\n", idx, idy, value);
}



extern "C"
void blur(std::vector<std::vector<float>>& matrix, std::vector<std::vector<float>>& result) {
    int h = matrix.size();
    int w = matrix[0].size();
    float* matrix_ptr = new float[w * h];
    float* result_ptr = new float[w * h];

    for (size_t i = 0; i < h; i++) {
        for (size_t j = 0; j < w; j++) {
            matrix_ptr[i * w + j] = matrix[i][j];
        }
    }

    float* matrix_gpu; float* result_gpu;
    int size = sizeof(float) * w * h;
    hipMalloc((void**) &matrix_gpu, size);
    hipMalloc((void**) &result_gpu, size);

    hipMemcpy(matrix_gpu, matrix_ptr, size, hipMemcpyHostToDevice);

    dim3 grid(ceil(h / 16.), ceil(w / 16.), 1);
    dim3 block(16, 16, 1);

    printf("h : %d, w : %d\n", h, w);
    blur_kernel<<<grid, block>>>(matrix_gpu, result_gpu, w, h, 3);
    hipDeviceSynchronize();

    hipMemcpy(result_ptr, result_gpu, size, hipMemcpyDeviceToHost);

    for(int i = 0; i < h; i++) {
        for(int j = 0; j < w; j++) {
            result[i].push_back(result_ptr[i * w + j]);
        } 
    }

    hipFree(matrix_gpu);
    hipFree(result_gpu);
    
    delete[] matrix_ptr;
    delete[] result_ptr;

}

__device__ int getIndex(Matrix M, int x, int y) {
    return M.n * x + y;
}

__global__ void matrixMultiply_kernel(Matrix A, Matrix B, Matrix C) {
    assert(A.n == B.m);
    int k = A.n;

    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int idy = blockDim.y * blockIdx.y + threadIdx.y;

    if(idx > -1 && idx < C.m && idy > -1 && idy < C.n) {
        float value = 0.;
        for(int i = 0; i < k; i++) {
            value += (A.elements[getIndex(A, idx, i)] * B.elements[getIndex(B, i, idy)]);
        }

        C.elements[getIndex(C, idx, idy)] = value;
        // printf("[IDX] : %d, [IDY] : %d, [VALUE] : %.2f\n", idx, idy, value);
    }


}

extern "C"
void matrixMultiply(Matrix A, Matrix B, Matrix C) {
    int BLOCK_WIDTH = 4;
    Matrix A_gpu(A.m, A.n);
    Matrix B_gpu(B.m, B.n);
    Matrix C_gpu(C.m, C.n);

    size_t size_A = sizeof(float) * A.m * A.n;
    size_t size_B = sizeof(float) * B.m * B.n;
    size_t size_C = sizeof(float) * C.m * C.n;

    hipMalloc((void **) &(A_gpu.elements), size_A);
    hipMalloc((void **) &(B_gpu.elements), size_B);
    hipMalloc((void **) &(C_gpu.elements), size_C);

    hipMemcpy(A_gpu.elements, A.elements, size_A, hipMemcpyHostToDevice);
    hipMemcpy(B_gpu.elements, B.elements, size_B, hipMemcpyHostToDevice);
    // hipMemcpy(C_gpu.elements, C.elements, size_C, hipMemcpyHostToDevice);

    dim3 grid(ceil((C.m + 0.) / BLOCK_WIDTH), ceil((C.n + 0.) / BLOCK_WIDTH));
    dim3 block(BLOCK_WIDTH, BLOCK_WIDTH);

    // printf("%d %d %d\n", grid.x, grid.y, grid.z);
    // printf("%d %d %d\n", block.x, block.y, block.z);

    matrixMultiply_kernel<<<grid, block>>>(A_gpu, B_gpu, C_gpu);

    hipMemcpy(C.elements, C_gpu.elements, size_C, hipMemcpyDeviceToHost);

    hipFree(A_gpu.elements);
    hipFree(B_gpu.elements);
    hipFree(C_gpu.elements);

}

extern "C"
void viewCudaDeviceInfo() {
    int num_device;
    hipGetDeviceCount(&num_device);

    hipDeviceProp_t prop;
    for(int i = 0; i < num_device; ++i) {
        hipGetDeviceProperties(&prop, i);
    }

    printf("[NUM_DEVICES] : %d, [MAX_THREAD_PER_BLOCK] : %d, [SHARE_MEMORY_SIZE] : %d\n", num_device, prop.maxThreadsPerBlock, prop.sharedMemPerBlock);
    printf("[MAX_BLOCK_X_SIZE] : %d, [MAX_BLOCK_Y_SIZE] : %d, [MAX_BLCOK_Z_SIZE] : %d\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
    
}

__device__ bool is_valid_conv1d(int index, int n) {
    if(index > -1 && index < n) {
        return true;
    }
    return false;
}

template<typename T>
__global__ void conv1d_kernel(T* v, T* result, T* m, int n, int k) {
    int step  = (k - 1) / 2;
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if(idx < n) {
        for(int i = -step; i <= step; ++i) {
            if(is_valid_conv1d(i + idx, n)) {
                result[idx] += v[i + idx] * m[step + i];
            }
        }
    }
}


template<typename T>
__global__ void conv1d_kernel_constant(T* v, T* result, int n, int k) {
    int step  = (k - 1) / 2;
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if(idx < n) {
        for(int i = -step; i <= step; ++i) {
            if(is_valid_conv1d(i + idx, n)) {
                result[idx] += v[i + idx] * weight[step + i];
            }
        }
    }
}

extern "C"
void conv1d(float* v, float* result, float* m, int n, int k) {
    assert(k % 2 == 1);
    float* v_gpu;
    float* result_gpu;
    float* m_gpu;
    int size = sizeof(float) * n;
    int size_mask = sizeof(float) * k;

    
    hipMalloc((void**) &v_gpu, size);
    hipMalloc((void**) &result_gpu, size);
    hipMalloc((void**) &m_gpu, size_mask);

    hipMemcpy(v_gpu, v, size, hipMemcpyHostToDevice);
    hipMemset(result_gpu, 0., size);
    hipMemcpyToSymbol(HIP_SYMBOL(weight), m, size_mask);
    hipMemcpy(m_gpu, m, size_mask, hipMemcpyHostToDevice);

    dim3 grid(ceil(n / 256.));
    dim3 block(256);
#ifdef DEBUG
    printf("[GRID X] : %d, [BLOCK X] : %d\n", grid.x, block.x);
#endif
    // conv1d_kernel<<<grid, block>>>(v_gpu, result_gpu, m_gpu, n, k);

    conv1d_kernel_constant<<<grid, block>>>(v_gpu, result_gpu, n, k);
    hipDeviceSynchronize();
    hipMemcpy(result, result_gpu, size, hipMemcpyDeviceToHost);
    
    hipFree(v_gpu);
    hipFree(result_gpu);
    hipFree(m_gpu);
}

template<typename T>
__global__ void mergeSort_kernel(T* v, T* temp, int n) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    for(int i = 2; i < 2 * n; i *= 2) {
        int len = i;
        if(n - idx < i) {
            len = n - idx;
        }


        if(idx % i == 0) {
            T* subA = &v[idx];
            int lenA = i / 2, k = 0;

            T* subB = &v[idx + lenA];
            int lenB = len - lenA, j = 0;

            int p = idx;
            while (/* condition */ k < lenA && j < lenB)
            {
                /* code */
                if(subA[k] < subB[j]) {
                    temp[p++] = subA[k];
                    k++;
                } else {
                    temp[p++] = subB[j];
                    j++;
                }
            }

            while(k < lenA) {
                temp[p++] = subA[k];
                k++;
            }
            
            while(j < lenB) {
                temp[p++] = subB[j];
                j++;
            }

            for(int m = idx; m < idx + len; m++) {
                v[m] = temp[m];
            }
        }
        __syncthreads();

    }
}

extern "C" 
void mergeSort(float* vector, int n) {
    float* vector_gpu;
    float* temp_gpu;
    int size = sizeof(float) * n;
    hipMalloc((void**) &vector_gpu, size);
    hipMalloc((void**) &temp_gpu, size);

    hipMemcpy(vector_gpu, vector, size, hipMemcpyHostToDevice);
    hipMemset(temp_gpu, 0, size);

    dim3 grid(ceil(n / 256.));
    dim3 block(ceil(256));

    mergeSort_kernel<<<grid, block>>>(vector_gpu, temp_gpu, n);
    hipDeviceSynchronize();

    hipMemcpy(vector, vector_gpu, size, hipMemcpyDeviceToHost);
    hipFree(vector_gpu);
    hipFree(temp_gpu);
}


template<typename T>
__global__ void vector_sum_kernel(T* vector, int n, T* result) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if(idx < n) {
        for(int i = 2 ; i < 2 * n; i *= 2) {
            if(idx % i == 0) {
                int index = idx + i / 2;
                if(index < n) {
                    vector[idx] += vector[index];
                }
            }
            __syncthreads();
        }
    }
}

extern "C"
void vector_sum(const float* vector, int n, float* result) {
    float* vector_gpu;
    float* v = new float[n];
    int size = sizeof(float) * n;
    hipMalloc((void **) &vector_gpu, size);
    hipMemcpy(vector_gpu, vector, size, hipMemcpyHostToDevice);

    dim3 grid(ceil( n / 256.));
    dim3 block(256);

#ifdef DEBUG
    printf("[GRID X] : %d, [BLOCK X] : %d\n", grid.x, block.x);
#endif

    vector_sum_kernel<<<grid, block>>>(vector_gpu, n, result);
    hipMemcpy(v, vector_gpu, size, hipMemcpyDeviceToHost);
#ifdef DEBUG
    printf("[");
    for(int i = 0; i < n; ++i) {
        printf("%.2f, ", v[i]);
    }
    printf("]\n");
#endif
    *result = v[0];
    hipFree(vector_gpu);
    delete[] v;
}



__global__ void BFS_kernel(int* V, int* E, int* F, int* visited, int num_v, int* result) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

#ifdef DEBUG
    // printf("[THREAD_ID] : %d\n", idx);
#endif

    if(idx < num_v) {
        if(F[idx]) {
            F[idx] = 0;
            visited[idx] = 1;
            // *result += nodes[idx].value;

            atomicAdd(result, nodes[idx].value);
            for(int i = V[idx]; i < V[idx + 1]; ++i) {
                if(!visited[E[i]]) {
                    atomicAdd(&F[E[i]], 1) ;
                }
            } 
        }
    }
}

__global__ void is_all_zero(int* vector, int n, int* result) {
    int idx =  blockDim.x * blockIdx.x + threadIdx.x;
    if(idx < n) {
        atomicAdd(result, vector[idx]);
    }
}

extern "C"
void BFS(std::vector<std::vector<int>> const& graph, std::vector<int> const& values, int* result) {
    int n = values.size();
    node* ns = new node[values.size()];
    std::vector<int> V, E; V.push_back(0);
    for(int i = 0 ; i < n; ++i) {
        ns[i].id = i;
        ns[i].value = values[i];
        for(auto const& id : graph[i]) {
            E.push_back(id);
        }
        V.push_back(E.size());
    }

    int* V_gpu;
    int* E_gpu;
    int* F_gpu;
    int* visited_gpu;
    int* flag;
    int* result_gpu;
    

    hipMalloc((void**) &V_gpu, sizeof(int) * V.size());
    hipMalloc((void**) &E_gpu, sizeof(int) * E.size());
    hipMallocManaged((void**) &F_gpu, sizeof(int) * n);
    hipMallocManaged((void**) &visited_gpu, sizeof(int) * n);
    hipMallocManaged((void**) &flag, sizeof(int));
    hipMallocManaged((void**) &result_gpu, sizeof(int));

    hipMemset(F_gpu, 0, sizeof(int) * n); F_gpu[0] = 1;
// #ifdef DEBUG
//     printf("%d\n", F_gpu[0]);
//     for(int i = 0; i < V.size(); i++) {
//         std::cout << V[i] << " ";
//     }
//     std::cout << std::endl;
//     for(int i = 0; i < E.size(); i++) {
//         std::cout << E[i] << " ";
//     }
// #endif
    hipMemset(visited_gpu, 0, sizeof(int) * n);
    hipMemcpy(V_gpu, V.data(), sizeof(int) * V.size(), hipMemcpyHostToDevice);
    hipMemcpy(E_gpu, E.data(), sizeof(int) * E.size(), hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(nodes), ns, sizeof(node) * n);

    dim3 grid(ceil(n / 256.));
    dim3 block(256);

#ifdef DEBUG
    printf("[GRID X] : %d, [BLOCK X] : %d\n", grid.x, block.x);
#endif
    *flag = 0;
    *result_gpu = 0;
    is_all_zero<<<grid, block>>>(F_gpu, n, flag);
#ifdef DEBUG
        printf("[FGPU] : %d\n", F_gpu[0]);
        printf("[F_TEMP] : %d\n", *flag);
#endif
    while(*flag != 0) {
#ifdef DEBUG
        printf("[F_TEMP] : %d\n", F_gpu[0]);
#endif
        BFS_kernel<<<grid, block>>>(V_gpu, E_gpu, F_gpu, visited_gpu, n, result_gpu);
        hipDeviceSynchronize();
        *flag = 0;
        is_all_zero<<<grid, block>>>(F_gpu, n, flag);
    }

    hipMemcpy(result, result_gpu, sizeof(int), hipMemcpyDeviceToHost);
    

    hipFree(V_gpu);
    hipFree(E_gpu);
    hipFree(F_gpu);
    hipFree(visited_gpu);
    hipFree(flag);
    


    delete[] ns; 
}

__global__ void vector_add_new_kernel(float* vector, float* result, int n) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
// #ifdef DEBUG
//     printf("[ID] : %d\n", idx);
// #endif
    if(idx < n) {
        // *result += vector[idx];
        atomicAdd(result, vector[idx]);
    }
}

extern "C"
void vector_add_new(float* vector, float* result, int n) {
    *result = 0;
    float* vector_gpu;
    float* result_gpu;

    hipMallocManaged((void**) &result_gpu, sizeof(float));
    *result_gpu = 0.;
    hipMalloc((void**) &vector_gpu, sizeof(float) * n);
    hipMemcpy(vector_gpu, vector, n * sizeof(float), hipMemcpyHostToDevice);

    dim3 grid(ceil(n / 256.)), block(256);
#ifdef DEBUG
    printf("[GRID X] : %d, [BLOCK X] : %d\n", grid.x, block.x);
#endif
    vector_add_new_kernel<<<grid, block>>>(vector_gpu, result_gpu, n);
    hipDeviceSynchronize();
    hipMemcpy(result, result_gpu, sizeof(float), hipMemcpyDeviceToHost);
    printf("[RESULT] : %.2f", *result);

    hipFree(vector_gpu);
    hipFree(result_gpu);
}


__global__ void realToComplex(float* vector, hipfftComplex* vector_complex, int n) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if(idx < n) {
        vector_complex[idx].x = vector[idx];
        vector_complex[idx].y = 0.0;
    }
}

__global__ void complexToCPU(hipfftComplex* vector_complex, float* real, float* image, int n) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if(idx < n) {
        real[idx] = vector_complex[idx].x;
        image[idx] = vector_complex[idx].y;
    }
}

extern "C"
void FFT1D(float* vector , float* real, float* image, int n) {
    float* vector_gpu, *real_gpu, *image_gpu;
    hipMalloc((void**) &vector_gpu, sizeof(float) * n);
    hipMalloc((void**) &real_gpu, sizeof(float) * n);
    hipMalloc((void**) &image_gpu, sizeof(float) * n);
    hipMemcpy(vector_gpu, vector, sizeof(float) * n, hipMemcpyHostToDevice);

    hipfftComplex *vector_complex, *fft_result;
    hipMalloc((void**) &vector_complex, sizeof(hipfftComplex) * n);
    hipMalloc((void**) &fft_result, sizeof(hipfftComplex) * n);
    dim3 grid(ceil(n / 256.));
    dim3 block(256);

    realToComplex<<<grid, block>>>(vector_gpu, vector_complex, n);
    hipfftHandle plan;
    hipfftPlan1d(&plan, n, HIPFFT_C2C, 1);
    hipfftExecC2C(plan, vector_complex, fft_result, HIPFFT_FORWARD);
    complexToCPU<<<grid, block>>>(fft_result, real_gpu, image_gpu, n);
    hipMemcpy(real, real_gpu, sizeof(float) * n, hipMemcpyDeviceToHost);
    hipMemcpy(image, image_gpu, sizeof(float) * n, hipMemcpyDeviceToHost);

    hipfftDestroy(plan);
    hipFree(vector_gpu);
    hipFree(real_gpu);
    hipFree(image_gpu);
    hipFree(vector_complex);
    hipFree(fft_result);
}

__global__ void complexMul(hipfftComplex* v1, hipfftComplex* v2, hipfftComplex* result, int n) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if(idx < n) {
        result[idx].x = (v1[idx].x * v2[idx].x - v1[idx].y * v2[idx].y) / n;
        result[idx].y = (v1[idx].x * v2[idx].y + v1[idx].y * v2[idx].x) / n;
    }
}

__global__ void complexToReal(hipfftComplex* complex, float* result, int n) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if(idx < n) {
        result[idx] = complex[idx].x / n;
    }
}


void FFTCONV1D(float* vector , float* kernel, float* result, int k , int n) {
    float* vector_gpu, *kernel_gpu, *result_gpu;
    hipMalloc((void**) &vector_gpu, sizeof(float) * n);
    hipMalloc((void**) &kernel_gpu, sizeof(float) * n);
    hipMalloc((void**) &result_gpu, sizeof(float) * n);

    hipMemcpy(vector_gpu, vector, sizeof(float) * n, hipMemcpyHostToDevice);
    hipMemset(kernel_gpu, 0., sizeof(float) * n);
    hipMemcpy(kernel_gpu, kernel, sizeof(float) * k, hipMemcpyHostToDevice);

    hipfftComplex *vector_fft;
    hipMalloc((void**) &vector_fft, sizeof(hipfftComplex) * n);

    hipfftComplex *kernel_fft, *cov_result_gpu;
    hipMalloc((void**) &kernel_fft, sizeof(hipfftComplex) * n);
    hipMalloc((void**) &cov_result_gpu, sizeof(hipfftComplex) * n);

    dim3 grid(ceil(n / 256.));
    dim3 block(256);

    hipfftHandle plan;
    hipfftPlan1d(&plan, n, HIPFFT_R2C, 1);
    hipfftExecR2C(plan, vector_gpu, vector_fft);
    hipfftExecR2C(plan, kernel_gpu, kernel_fft);



    complexMul<<<grid, block>>>(vector_fft, kernel_fft, cov_result_gpu, n);
    hipfftHandle plan_I;
    hipfftPlan1d(&plan_I, n, HIPFFT_C2R, 1);
    hipfftExecC2R(plan_I, cov_result_gpu, result_gpu);

    hipMemcpy(result, result_gpu, sizeof(float) * n, hipMemcpyDeviceToHost);

    hipfftDestroy(plan);
    hipfftDestroy(plan_I);
    hipFree(vector_gpu);
    hipFree(kernel_gpu);
    hipFree(result_gpu);

    hipFree(vector_fft);

    hipFree(kernel_fft);
    hipFree(cov_result_gpu);

}

__global__ void MatrixElementMult_kernel(float* m1, float* m2, float* result, int m, int n) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int idy = blockDim.y * blockIdx.y + threadIdx.y;

    if(idx < m && idy < n) {
        int index = idx * n + idy;
        result[index] = m1[index] * m2[index];
    }
}


extern "C"
void MatrixElementMult(float* m1, float* m2, float* result, int m, int n) {
    float *m1_gpu, *m2_gpu, *result_gpu;
    int size = sizeof(float) * m * n;
    hipMalloc((void**) &m1_gpu, size);
    hipMalloc((void**) &m2_gpu, size);
    hipMalloc((void**) &result_gpu, size);
    hipMemcpy(m1_gpu, m1, size, hipMemcpyHostToDevice);
    hipMemcpy(m2_gpu, m2, size, hipMemcpyHostToDevice);
    hipMemset(result_gpu, 0., size);
    dim3 grid(ceil(m / 16.), ceil(n / 16.));
    dim3 block(16, 16);

#ifdef DEBUG
    printf("[GRID X] : %d, [BLOCK X] : %d\n", grid.x, block.x);
#endif

    MatrixElementMult_kernel<<<grid, block>>>(m1_gpu, m2_gpu, result_gpu, m, n);
    hipDeviceSynchronize();
    hipMemcpy(result, result_gpu, size, hipMemcpyDeviceToHost);

    hipFree(m1_gpu);
    hipFree(m2_gpu);
    hipFree(result_gpu);
}


__device__ __host__ void ComplexMultFunction(hipfftComplex& input1, hipfftComplex& input2, hipfftComplex& output, int n, bool need_scale) {
    if(need_scale) {
        output.x = (input1.x * input2.x - input1.y * input2.y) / n;
        output.y = (input1.x * input2.y + input1.y * input2.x) / n;
    }else{
        output.x = input1.x * input2.x - input1.y * input2.y;
        output.y = input1.x * input2.y + input1.y * input2.x;
    }
}

__global__ void FFTCONV2D_kernel(hipfftComplex* matrix, hipfftComplex* kernel, hipfftComplex* result, int m, int n, bool need_scale) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int idy = blockDim.y * blockIdx.y + threadIdx.y;

    int index = idy * m + idx;

    if(idx < m && idy < n) {
        ComplexMultFunction(matrix[index], kernel[index], result[index], m * n, need_scale);
    }
    // printf("[IDX] : %d, [IDY] : %d", idx, idy);
}

extern "C"
void FFTCONV2D(float* m1, float* m2 , float* result, int m, int n, int k) {
    bool scale = true;
    assert(k < m && k < n);
    float *m1_gpu, *m2_gpu, *result_gpu;
    int size = sizeof(float) * m * n;
    hipMallocManaged((void **) &m1_gpu, size);
    hipMallocManaged((void **) &m2_gpu, size);
    hipMallocManaged((void **) &result_gpu, size);

    hipMemset(m2_gpu, 0, size);
    for(int i = 0; i < k; ++i) {
        for(int j = 0; j < k; ++j) {
            m2_gpu[i * n + j] = m2[i * k + j];
        }
    }

    hipMemcpy(m1_gpu, m1, size, hipMemcpyHostToDevice);

    int size_fft = sizeof(hipfftComplex) * m * n; 
    hipfftComplex *m1_fft, *m2_fft, *result_fft;
    hipMallocManaged((void **) &m1_fft, size_fft);
    hipMallocManaged((void **) &m2_fft, size_fft);
    hipMallocManaged((void **) &result_fft, size_fft);

    hipfftHandle fftPlan;
    hipfftPlan2d(&fftPlan, m, n, HIPFFT_R2C);
    hipfftExecR2C(fftPlan, m1_gpu, m1_fft);
    hipfftExecR2C(fftPlan, m2_gpu, m2_fft);

    hipError_t err = hipGetLastError();
    

    hipFree(m1_gpu);
    hipFree(m2_gpu);
    hipfftDestroy(fftPlan);

    dim3 grid(ceil(m / 32.), ceil(n / 32.));
    dim3 block(32, 32);

#ifdef DEBUG
    printf("[GRID X] : %d, [BLOCK X] : %d, [GRID Y] : %d, [BLOCK Y] : %d\n",grid.x, block.x, grid.y, block.y);
#endif

    FFTCONV2D_kernel<<<grid, block>>>(m1_fft, m2_fft, result_fft, m, n, scale);
    
    if (err != hipSuccess) {
        printf("CUDA Error: %s\n", hipGetErrorString(err));
        viewCudaDeviceInfo();
        exit(-1);
    } 
    
    hipDeviceSynchronize();
    
    hipFree(m1_fft);
    hipFree(m2_fft);

    hipfftHandle ifftPlan;
    hipfftPlan2d(&ifftPlan, m, n, HIPFFT_C2R);
    hipfftExecC2R(ifftPlan, result_fft, result_gpu);
    hipFree(result_fft);
    hipfftDestroy(ifftPlan);

    hipMemcpy(result, result_gpu, size, hipMemcpyDeviceToHost);
    hipFree(result_gpu);
}





